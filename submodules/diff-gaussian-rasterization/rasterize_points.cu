#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include <math.h>
#include <torch/extension.h>
#include <cstdio>
#include <sstream>
#include <iostream>
#include <tuple>
#include <stdio.h>
#include <hip/hip_runtime_api.h>
#include <memory>
#include "cuda_rasterizer/config.h"
#include "cuda_rasterizer/rasterizer.h"
#include <fstream>
#include <string>
#include <functional>

std::function<char*(size_t N)> resizeFunctional(torch::Tensor& t) {
    auto lambda = [&t](size_t N) {
        t.resize_({(long long)N});
		return reinterpret_cast<char*>(t.contiguous().data_ptr());
    };
    return lambda;
}

std::tuple<int, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
RasterizeGaussiansCUDA(
	const torch::Tensor& background = torch::empty({0}, torch::kHalf),
	const torch::Tensor& means3D = torch::empty({0}, torch::kHalf),
    const torch::Tensor& colors = torch::empty({0}, torch::kHalf),
    const torch::Tensor& opacity = torch::empty({0}, torch::kHalf),
	const torch::Tensor& scales = torch::empty({0}, torch::kHalf),
	const torch::Tensor& rotations = torch::empty({0}, torch::kHalf),
	const float scale_modifier = 0.0,
	const torch::Tensor& cov3D_precomp = torch::empty({0}, torch::kHalf),
	const torch::Tensor& viewmatrix = torch::empty({0}, torch::kHalf),
	const torch::Tensor& projmatrix = torch::empty({0}, torch::kHalf),
	const float tan_fovx = 0.0, 
	const float tan_fovy = 0.0,
    const int image_height = 0,
    const int image_width = 0,
	const torch::Tensor& sh = torch::empty({0}, torch::kHalf),
	const int degree = 0,
	const torch::Tensor& campos = torch::empty({0}, torch::kHalf),
	const bool prefiltered = false,
	const bool debug = false)
{
  if (means3D.ndimension() != 2 || means3D.size(1) != 3) {
    AT_ERROR("means3D must have dimensions (num_points, 3)");
  }
  
  const int P = means3D.size(0);
  const int H = image_height;
  const int W = image_width;

  auto int_opts = means3D.options().dtype(torch::kInt32);
  auto float_opts = means3D.options().dtype(torch::kHalf);

  torch::Tensor out_color = torch::full({NUM_CHANNELS, H, W}, 0.0, float_opts);
  torch::Tensor radii = torch::full({P}, 0, means3D.options().dtype(torch::kInt32));
  
  torch::Device device(torch::kCUDA);
  torch::TensorOptions options(torch::kByte);
  torch::Tensor geomBuffer = torch::empty({0}, options.device(device));
  torch::Tensor binningBuffer = torch::empty({0}, options.device(device));
  torch::Tensor imgBuffer = torch::empty({0}, options.device(device));
  std::function<char*(size_t)> geomFunc = resizeFunctional(geomBuffer);
  std::function<char*(size_t)> binningFunc = resizeFunctional(binningBuffer);
  std::function<char*(size_t)> imgFunc = resizeFunctional(imgBuffer);
  
  int rendered = 0;
  if(P != 0)
  {
	  int M = 0;
	  if(sh.size(0) != 0)
	  {
		M = sh.size(1);
      }

	  rendered = CudaRasterizer::Rasterizer::forward(
	    geomFunc,
		binningFunc,
		imgFunc,
	    P, degree, M,
		background.contiguous().data<__half>(),
		W, H,
		means3D.contiguous().data<__half>(),
		sh.contiguous().data_ptr<__half>(),
		colors.contiguous().data<__half>(), 
		opacity.contiguous().data<__half>(), 
		scales.contiguous().data_ptr<__half>(),
		scale_modifier,
		rotations.contiguous().data_ptr<__half>(),
		cov3D_precomp.contiguous().data<__half>(), 
		viewmatrix.contiguous().data<__half>(), 
		projmatrix.contiguous().data<__half>(),
		campos.contiguous().data<__half>(),
		tan_fovx,
		tan_fovy,
		prefiltered,
		out_color.contiguous().data<__half>(),
		radii.contiguous().data<int>(),
		debug);
  }
  return std::make_tuple(rendered, out_color, radii, geomBuffer, binningBuffer, imgBuffer);
}

std::tuple<torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor, torch::Tensor>
 RasterizeGaussiansBackwardCUDA(
 	const torch::Tensor& background = torch::empty({0}, torch::kHalf),
	const torch::Tensor& means3D = torch::empty({0}, torch::kHalf),
	const torch::Tensor& radii = torch::empty({0}, torch::kHalf),
    const torch::Tensor& colors = torch::empty({0}, torch::kHalf),
	const torch::Tensor& scales = torch::empty({0}, torch::kHalf),
	const torch::Tensor& rotations = torch::empty({0}, torch::kHalf),
	const float scale_modifier = 0.0f,
	const torch::Tensor& cov3D_precomp = torch::empty({0}, torch::kHalf),
	const torch::Tensor& viewmatrix = torch::empty({0}, torch::kHalf),
    const torch::Tensor& projmatrix = torch::empty({0}, torch::kHalf),
	const float tan_fovx = 0.0f,
	const float tan_fovy = 0.0f,
    const torch::Tensor& dL_dout_color = torch::empty({0}, torch::kHalf),
	const torch::Tensor& sh = torch::empty({0}, torch::kHalf),
	const int degree =0,
	const torch::Tensor& campos = torch::empty({0}, torch::kHalf),
	const torch::Tensor& geomBuffer = torch::empty({0}, torch::kHalf),
	const int R = 0,
	const torch::Tensor& binningBuffer = torch::empty({0}, torch::kHalf),
	const torch::Tensor& imageBuffer = torch::empty({0}, torch::kHalf),
	const bool debug=false) 
{
  const int P = means3D.size(0);
  const int H = dL_dout_color.size(1);
  const int W = dL_dout_color.size(2);
  
  int M = 0;
  if(sh.size(0) != 0)
  {	
	M = sh.size(1);
  }

  torch::Tensor dL_dmeans3D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dmeans2D = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_dcolors = torch::zeros({P, NUM_CHANNELS}, means3D.options());
  torch::Tensor dL_dconic = torch::zeros({P, 2, 2}, means3D.options());
  torch::Tensor dL_dopacity = torch::zeros({P, 1}, means3D.options());
  torch::Tensor dL_dcov3D = torch::zeros({P, 6}, means3D.options());
  torch::Tensor dL_dsh = torch::zeros({P, M, 3}, means3D.options());
  torch::Tensor dL_dscales = torch::zeros({P, 3}, means3D.options());
  torch::Tensor dL_drotations = torch::zeros({P, 4}, means3D.options());
  
  if(P != 0)
  {  
	  CudaRasterizer::Rasterizer::backward(P, degree, M, R,
	  background.contiguous().data<__half>(),
	  W, H, 
	  means3D.contiguous().data<__half>(),		//
	  sh.contiguous().data<__half>(),
	  colors.contiguous().data<__half>(),
	  scales.data_ptr<__half>(),
	  scale_modifier,
	  rotations.data_ptr<__half>(),
	  cov3D_precomp.contiguous().data<__half>(),
	  viewmatrix.contiguous().data<__half>(),		//
	  projmatrix.contiguous().data<__half>(),
	  campos.contiguous().data<__half>(),
	  tan_fovx,
	  tan_fovy,
	  radii.contiguous().data<int>(),
	  reinterpret_cast<char*>(geomBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(binningBuffer.contiguous().data_ptr()),
	  reinterpret_cast<char*>(imageBuffer.contiguous().data_ptr()),
	  dL_dout_color.contiguous().data<__half>(),
	  dL_dmeans2D.contiguous().data<__half>(),
	  dL_dconic.contiguous().data<__half>(),  
	  dL_dopacity.contiguous().data<__half>(),
	  dL_dcolors.contiguous().data<__half>(),
	  dL_dmeans3D.contiguous().data<__half>(),
	  dL_dcov3D.contiguous().data<__half>(),
	  dL_dsh.contiguous().data<__half>(),
	  dL_dscales.contiguous().data<__half>(),
	  dL_drotations.contiguous().data<__half>(),
	  debug);
  }

  return std::make_tuple(dL_dmeans2D, dL_dcolors, dL_dopacity, dL_dmeans3D, dL_dcov3D, dL_dsh, dL_dscales, dL_drotations);
}

torch::Tensor markVisible(
		torch::Tensor& means3D,
		torch::Tensor& viewmatrix,
		torch::Tensor& projmatrix)
{ 
  const int P = means3D.size(0);
  
  torch::Tensor present = torch::full({P}, false, means3D.options().dtype(at::kBool));
 
  if(P != 0)
  {
	CudaRasterizer::Rasterizer::markVisible(P,
		means3D.contiguous().data<__half>(),
		viewmatrix.contiguous().data<__half>(),
		projmatrix.contiguous().data<__half>(),
		present.contiguous().data<bool>());
  }
  
  return present;
}